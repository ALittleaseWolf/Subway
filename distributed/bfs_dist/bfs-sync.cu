#include "hip/hip_runtime.h"
#include "../../shared/timer.hpp"


#include "../../shared/subgraph.cuh"
#include "../../shared/partitioner.cuh"
#include "../../shared/subgraph_generator.cuh"
#include "../../shared/gpu_error_check.cuh"
#include "../../shared/gpu_kernels.cuh"
#include "../../shared/subway_utilities.hpp"
#include "bfs_dis.h"
void bfs_sync(Graph<OutEdge> G, ArgumentParser arguments, uint graph_value[])
{
    hipFree(0);
    
    // cout<< arguments.input << endl;
    Timer timer;
    timer.Start();
    Graph<OutEdge> graph;
    graph.ReadGraphFromGraph(G);
    for(uint i=0; i<graph.num_nodes;i++)
    {
        graph.value[i] = graph_value[i];
        if(graph_value[i] != DIST_INFINITY)
        {
            graph.label2[i] = true;
        }
        graph.label1[i] = false;
    }
    graph.value[arguments.sourceNode] = 0;
	graph.label1[arguments.sourceNode] = false;
	graph.label2[arguments.sourceNode] = true;

    float readtime = timer.Finish();
    cout << "Graph Reading fininshed in " << readtime/1000 << "(s).\n";

    gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_label2, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));

    // subway 快速子图生成
    Subgraph<OutEdge> subgraph(graph.num_nodes, graph.num_edges);

    SubgraphGenerator<OutEdge> subgen(graph);

    subgen.generate(graph, subgraph);

    Partitioner<OutEdge> partitioner;

// 没好好分析这里
    timer.Start();
    uint itr = 0;
    while( subgraph.numActiveNodes >0)
    {
        itr ++;
        partitioner.partition(subgraph, subgraph.numActiveNodes);

        for(int i=0;i<partitioner.numPartitions;i++)
        {
            hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			hipDeviceSynchronize();

            moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);

			bfs_kernel<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
													partitioner.fromNode[i],
													partitioner.fromEdge[i],
													subgraph.d_activeNodes,
													subgraph.d_activeNodesPointer,
													subgraph.d_activeEdgeList,
													graph.d_outDegree,
													graph.d_value, 
													//d_finished,
													graph.d_label1,
													graph.d_label2);

			hipDeviceSynchronize();
			gpuErrorcheck( hipPeekAtLastError() );	
        }

        subgen.generate(graph, subgraph);
    }

    float runtime = timer.Finish();
	cout << "Processing finished in " << runtime/1000 << " (s).\n";
	
	cout << "Number of iterations = " << itr << endl;
	gpuErrorcheck(hipMemcpy(graph.value, graph.d_value, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost));
	
	utilities::PrintResults(graph.value, min(30, graph.num_nodes));
	
    for(uint i=0;i<graph.num_nodes;i++)
        graph_value[i] = graph_value[i]<graph.value[i]? graph_value[i]:graph.value[i];

    gpuErrorcheck(hipFree(graph.d_outDegree));
    gpuErrorcheck(hipFree(graph.d_value));
    gpuErrorcheck(hipFree(graph.d_label1));
    gpuErrorcheck(hipFree(graph.d_label2));
    gpuErrorcheck(hipHostFree(graph.edgeList));
	// if(arguments.hasOutput)
	// 	utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);

}
